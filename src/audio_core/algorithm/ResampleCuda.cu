#include "hip/hip_runtime.h"
//to understand data types: http://michas.eu/blog/c_ints.php?lang=en
#include <hip/hip_runtime.h>
#include "" 
#include <array>
#include "hip/hip_runtime.h"
#include <cmath>
#include "common/common_types.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/device_free.h> // place holder for testing
#include <thrust/device_malloc.h> // place holder for testing


//global vars are used to force variables into l2 cache when possible; also to prevent variables from deallocating whenever possible

__constant__ s16 lutTd[512];

__global__ void ResampleKernel(s32* output, s32* fraction) {

    std::size_t i = threadIdx.x;
        
    const std::size_t lut_index{ (static_cast<std::size_t>(fraction[i + 1]) >> 8) * 4 }; //fraction is s32, lut_index is size_t


    const s16 l0 = lutTd[lut_index + 0]; //faster this way
    const s16 l1 = lutTd[lut_index + 1];
    const s16 l2 = lutTd[lut_index + 2];
    const s16 l3 = lutTd[lut_index + 3];

    const s32 s0 = fraction[(fraction[i + fraction[0] + 1] + 0 + fraction[0] * 2 + 1)];
    const s32 s1 = fraction[(fraction[i + fraction[0] + 1] + 0 + fraction[0] * 2 + 2)];
    const s32 s2 = fraction[(fraction[i + fraction[0] + 1] + 0 + fraction[0] * 2 + 3)];
    const s32 s3 = fraction[(fraction[i + fraction[0] + 1] + 0 + fraction[0] * 2 + 4)];


    output[i] = (l0 * s0 + l1 * s1 + l2 * s2 + l3 * s3) >> 15;
}

thrust::device_vector <s32> postFractiond; 
thrust::device_vector <s32> outD; 
thrust::host_vector<s32> postFraction; 

extern "C" void ResampleCuda(std::size_t sample_count, s32 * fraction, s32 * output, const s32* input, s32 pitch, const std::array<s16, 512> lut) {
    
    hipSetDeviceFlags(hipDeviceLmemResizeToMax); // doing good?
    

    hipMemcpyToSymbolAsync(HIP_SYMBOL(lutTd), &lut, sizeof(s16) * (512), 0, hipMemcpyHostToDevice); //constant memory filled with lut curve values

    
    /* 
    I put together in postFraction in this order for memcpy speed: sample_count [populates index 0], fraction
     values [size of sample_count], index [size of sample_count], input [size of sample_count + 3]   
    */
    
    postFraction.resize(sample_count * 3 + 4); 
    //postFractiond.resize(sample_count * 3 + 4);

    postFraction[1] = *fraction;
    postFraction[sample_count + 1] = 0;
    postFraction[0] = sample_count;

    thrust::copy(input,input+sample_count+3, postFraction.begin()+sample_count * 2 + 1); // copy all 'input' array values

    for (std::size_t i = 1; i < sample_count + 1;
         i++) { 

        postFraction[i + 1] = postFraction[i] + pitch;

        postFraction[i + sample_count + 1] =
            postFraction[i + sample_count] + (postFraction[i + 1] >> 15);

        postFraction[i + 1] &= 0x7fff;
    }
   
    postFractiond = postFraction;

    s32* postFractionP = thrust::raw_pointer_cast(postFractiond.data()); 

    outD.resize(sample_count); //resize premade vector that is in global l2 cache

    s32* outDP = thrust::raw_pointer_cast(outD.data());

    hipDeviceSynchronize(); // sync up all thread

   // hipMemPrefetchAsync(postFractionP, sizeof(s32) * (sample_count * 3 + 4), NULL);

   // hipMemPrefetchAsync(outDP, sizeof(s32) * (sample_count), NULL);

    ResampleKernel <<<1, sample_count>>>(outDP, postFractionP);

    /* KERNEL IS SUPPOSED TO EMULATE THE OPERATION BELOW
    //for (std::size_t i = 0; i < sample_count; i++) {

        const std::size_t lut_index{ (static_cast<std::size_t>(postFraction[i]) >> 8) * 4 }; //fraction is s32, lut_index is size_t

        const s16 l0 = lutH[lut_index + 0]; // s16
        const s16 l1 = lutH[lut_index + 1]; // s16
        const s16 l2 = lutH[lut_index + 2]; // s16
        const s16 l3 = lutH[lut_index + 3]; // s16

        const s32 s0 = (inputH[index[i] + 0]); //s32
        const s32 s1 = (inputH[index[i] + 1]); // s32  
        const s32 s2 = (inputH[index[i] + 2]); // s32
        const s32 s3 = (inputH[index[i] + 3]); // s32

        out[i] = (l0 * s0 + l1 * s1 + l2 * s2 + l3 * s3) >> 15; // output is s32
    */
    
    thrust::copy(outD.begin(), outD.end(),
                 output); // for now thrust::copy seems like the fastest copy operation

}

//burgy kirby ;')
